#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "Intersection.h"
#include "Timer.h"
#include "Tuple.h"
#include "Constants.h"
#include "Ray.h"
#include "Shape.h"
#include "Sphere.h"
//#include "Quad.h"
#include "Utils.h"
#include "Types.h"
#include "Material.h"
#include "Camera.h"
#include "World.h"
#include "kernel.h"
#include "Shading.h"
#include "Light.h"

#include "KernelRandom.h"

#include <math.h>

#include <stdio.h>

#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <algorithm>
#include <vector>

struct Matrix {
    int32_t width;
    int32_t height;
    double* elements;
};

struct Viewport {
    double scale;
    double fov;
    double imageAspectRatio;
    double width;
    double height;
};

constexpr int32_t objectCount = 2;
constexpr int32_t lightCount = 1;
constexpr int32_t materialCount = 2;

Payload* payload = nullptr;

Shape** objects[objectCount];
Light** lights[lightCount];
Material** materials[materialCount];
Shape** quad = nullptr;

#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

CUDA_DEVICE void writePixel(uint8_t* pixelBuffer, int32_t index, const Tuple& pixelColor) {
    pixelBuffer[index] = 256 * std::clamp(pixelColor.x(), 0.0, 0.999);
    pixelBuffer[index + 1] = 256 * std::clamp(pixelColor.y(), 0.0, 0.999);
    pixelBuffer[index + 2] = 256 * std::clamp(pixelColor.z(), 0.0, 0.999);
}

CUDA_GLOBAL void createQuad(Shape** object, Matrix4 transform) {
    //(*object) = new Quad();
    ////(*object)->setTransformation(transform);
    //(*object)->material = new Material();
}

CUDA_GLOBAL void createObject(Shape** object, Tuple origin, double radius) {
    // It is necessary to create object representing a function
    // directly in global memory of the GPU device for virtual
    // functions to work correctly, i.e. virtual function table
    // HAS to be on GPU as well.
    //auto index = threadIdx.x;
    (*object) = new Sphere(origin, radius);
    //(*object)->setTransformation(transform);
    (*object)->material = new Material();
}

//CUDA_GLOBAL void createObject(Shape** object, Tuple origin, double radius, Material** material, Matrix4 transform) {
//    // It is necessary to create object representing a function
//    // directly in global memory of the GPU device for virtual
//    // functions to work correctly, i.e. virtual function table
//    // HAS to be on GPU as well.
//    //auto index = threadIdx.x;
//    (*object) = new Sphere(origin, radius);
//    //(*object)->setTransformation(transform);
//    (*object)->material = new Material();
//}

CUDA_GLOBAL void createLight(Light** light, Tuple inPosition, Tuple inIntensity, Matrix4 transform) {
    // It is necessary to create object representing a function
    // directly in global memory of the GPU device for virtual
    // functions to work correctly, i.e. virtual function table
    // HAS to be on GPU as well.
    (*light) = new Light(inPosition, inIntensity);
    //(*light)->transform(transform);
}

CUDA_GLOBAL void createMaterial(Material** material) {
    // It is necessary to create object representing a function
    // directly in global memory of the GPU device for virtual
    // functions to work correctly, i.e. virtual function table
    // HAS to be on GPU as well.
    (*material) = new Material();
}

template<typename T>
CUDA_GLOBAL void deleteObject(T** object) {
    delete (*object);
}

CUDA_GLOBAL void fillBufferKernel(int32_t width, int32_t height, Payload* payload) {
    int32_t row = threadIdx.y + blockIdx.y * blockDim.y;
    int32_t column = threadIdx.x + blockIdx.x * blockDim.x;
    int32_t index = row * width + column;
    
    //row = 159;
    //column = 192;

    //auto viewport = payload->viewport;

    Tuple defaultColor = Color::skyBlue;
    Tuple pixelColor = defaultColor;

    const int32_t samplesPerPixel = 1;

    for (int i = 0; i < samplesPerPixel; i++) {
        //hiprandState state;
        //hiprand_init((unsigned long long)clock() + column, 0, 0, &state);

        double rx = 0.0; // hiprand_uniform_double(&state);
        double ry = 0.0; // hiprand_uniform_double(&state);

        //auto x = (viewport->height * (column + 0.5 + rx) / width - 1) * viewport->imageAspectRatio * viewport->scale;
        //auto y = (1.0 - viewport->height * (row + 0.5 + ry) / height) * viewport->scale;
        auto x = (static_cast<double>(column) + rx) / (width - 1);
        auto y = (static_cast<double>(row) + ry) / (height - 1);
        
        auto ray = payload->camera->getRay(x, y);

        auto hitInfo = colorAt(payload->world, ray);

        if (hitInfo.bHit) {
            pixelColor = hitInfo.surface;

            auto scatter = Color::white;

            for (auto i = 0; i < 5; i++) {
                scatter = scatter * computeReflectionAndRefraction(hitInfo, payload->world);
            }

            pixelColor += scatter;
        }
    }

    writePixel(payload->pixelBuffer, index * 3, (pixelColor ) / samplesPerPixel);
}

void queryDeviceProperties() {
    int32_t deviceIndex = 0;
    hipDeviceProp_t devicePro;
    hipGetDeviceProperties(&devicePro, deviceIndex);

    std::cout << "使用的GPU device：" << deviceIndex << ": " << devicePro.name << std::endl;
    std::cout << "SM的数量：" << devicePro.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devicePro.sharedMemPerBlock / 1024.0 << "KB\n";
    std::cout << "每个SM的最大线程块数：" << devicePro.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "每个线程块的最大线程数：" << devicePro.maxThreadsPerBlock << std::endl;
    std::cout << "每个SM的最大线程数：" << devicePro.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个SM的最大线程束数：" << devicePro.warpSize << std::endl;
}

void cleanup() {
    //deleteObject<<<1, 1>>>(quad);

    gpuErrorCheck(hipFree(payload->world));

    for (auto i = 0; i < materialCount; i++) {
        deleteObject<<<1, 1>>>(materials[i]);
    }

    for (auto i = 0; i < lightCount; i++) {
        deleteObject<<<1, 1>>>(lights[i]);
    }

    for (auto i = 0; i < objectCount; i++) {
        deleteObject<<<1, 1>>>(objects[i]);
    }

    gpuErrorCheck(hipDeviceSynchronize());

    for (auto i = 0; i < objectCount; i++) {
        gpuErrorCheck(hipFree(objects[i]));
    }

    gpuErrorCheck(hipFree(payload->viewport));
    gpuErrorCheck(hipFree(payload->pixelBuffer));
}

int32_t size = 0;
std::shared_ptr<ImageData> imageData;

void initialize(int32_t width, int32_t height) {
    // Choose which GPU to run on, change this on a multi-GPU system.
    gpuErrorCheck(hipSetDevice(0));

    gpuErrorCheck(hipMallocManaged((void**)&payload, sizeof(Payload)));

    gpuErrorCheck(hipMallocManaged((void**)&payload->pixelBuffer, width * height * 3 * sizeof(uint8_t)));

    gpuErrorCheck(hipMallocManaged((void**)&payload->viewport, sizeof(Viewport)));

    payload->viewport->fov = 90.0;
    payload->viewport->scale = std::tan(Math::radians(payload->viewport->fov / 2));

    payload->viewport->imageAspectRatio = static_cast<double>(width) / height;

    payload->viewport->height = 2.0 * payload->viewport->scale;
    payload->viewport->width = payload->viewport->height * payload->viewport->imageAspectRatio;

    gpuErrorCheck(hipMallocManaged((void**)&payload->camera, sizeof(Camera)));

    payload->camera->init(width, height);
    payload->camera->computeParameters();

    for (auto i = 0; i < materialCount; i++) {
        gpuErrorCheck(hipMallocManaged((void**)&materials[i], sizeof(Material**)));
    }

    createMaterial<<<1, 1>>>(materials[0]);
    createMaterial<<<1, 1>>>(materials[1]);

    for (auto i = 0; i < objectCount; i++) {
        gpuErrorCheck(hipMallocManaged((void**)&objects[i], sizeof(Shape**)));
    }

    Tuple origins[objectCount];

    origins[0] = point(-1.5, 0.0, -2.0);
    origins[1] = point(1.5, 0.0, -2.0);

    double radiuses[objectCount] = { 1.0, 1.0 };

    createObject<<<1, 1>>>(objects[0], origins[0], radiuses[0]);
    createObject<<<1, 1>>>(objects[1], origins[1], radiuses[1]);
    //createObject<<<1, 1>>>(objects[0], origins[0], radiuses[0], materials[0], Matrix4());
    //createObject<<<1, 1>>>(objects[1], origins[1], radiuses[1], materials[1], Matrix4());
    //createObject<<<1, 1>>>(objects[0], point( 1.5, 0.0, -3.0), 1.0, *materials[0]);
    //createObject<<<1, 1>>> (objects[2], point(-4.0, 0.0, -3.0), 1.0);
    //createObject<<<1, 1>>> (objects[3], point( 3.0, 0.0, -3.0), 1.0);

    for (auto i = 0; i < lightCount; i++) {
        gpuErrorCheck(hipMallocManaged((void**)&lights[i], sizeof(Light**)));
    }

    createLight<<<1, 1>>>(lights[0], point(0.0, 1.0, 0.0), Tuple(1.0, 1.0, 1.0), Matrix4());

    //gpuErrorCheck(hipMallocManaged((void**)&quad, sizeof(Quad**)));

    //createQuad<<<1, 1>>>(quad, translate(0.0, -2.0, -6.0) * scaling(5.0, 1.0, 5.0));

    gpuErrorCheck(hipDeviceSynchronize());

    gpuErrorCheck(hipMallocManaged((void**)&payload->world, sizeof(World)));

    for (auto i = 0; i < objectCount; i++) {
        payload->world->addObject(*objects[i]);
    }

    //payload->world->addObject(*quad);

    for (auto i = 0; i < lightCount; i++) {
        payload->world->addLight(*lights[i]);
    }

    size = width * height * 3;
    imageData = std::make_shared<ImageData>();
    imageData->data = new uint8_t[size];
}

//ImageData* launch(int32_t width, int32_t height) {
//    //queryDeviceProperties();
//
//    //Timer timer;
//
//    initialize(width, height);
//
//    dim3 blockSize(32, 32);
//    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
//        (height + blockSize.y - 1) / blockSize.y);
//
//    fillBufferKernel<<<gridSize, blockSize>>>(width, height, payload);
//
//    gpuErrorCheck(hipDeviceSynchronize());
//
//    //timer.stop();
//
//    writeToPPM("render.ppm", width, height, payload->pixelBuffer);
//
//    imageData->width = width;
//    imageData->height = height;
//    imageData->data = payload->pixelBuffer;
//    imageData->channels = 3;
//    imageData->size = size;
//
//    return imageData.get();
//}

int main() {
    //queryDeviceProperties();

    constexpr int32_t width = 480;
    constexpr int32_t height = 320;

    initialize(width, height);

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
        (height + blockSize.y - 1) / blockSize.y);

    Timer timer;

    fillBufferKernel<<<gridSize, blockSize >>>(width, height, payload);

    gpuErrorCheck(hipDeviceSynchronize());

    timer.stop();

    writeToPPM("render.ppm", width, height, payload->pixelBuffer);

    return 0;
}