#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "Intersection.h"
#include "Timer.h"
#include "Tuple.h"
#include "Constants.h"
#include "Ray.h"
//#include "Sphere.h"
#include "Shape.h"
#include "Utils.h"
#include "Types.h"
#include "Material.h"
#include "Camera.h"
#include "World.h"

#include "KernelRandom.h"

#include <math.h>

#include <stdio.h>

#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <algorithm>
#include <vector>

struct Matrix {
    int32_t width;
    int32_t height;
    double* elements;
};

struct Viewport {
    double scale;
    double fov;
    double imageAspectRatio;
    double width;
    double height;
};

struct Payload {
    Intersection* intersections;
    World* world;
    int32_t objectCount = 2;
    Viewport* viewport;
    Tuple* pixelBuffer;
    Camera* camera;
};

class Sphere : public Shape {
public:
    CUDA_HOST_DEVICE Sphere()
    : origin({ 0.0, 0.0, 0.0 }), radius(1.0) {}

    CUDA_HOST_DEVICE Sphere(const Tuple& inOrigin, double inRadius = 1.0) 
    : origin(inOrigin), radius(inRadius) {}

    CUDA_HOST_DEVICE void foo() override {}

    inline CUDA_HOST_DEVICE Tuple normalAt(const Tuple& position) const override {
        auto normal = (position - origin);
        return  normal.normalize();
    }

    inline CUDA_HOST_DEVICE void intersect(const Ray& ray, Intersection* intersections) override {
        auto oc = (ray.origin - origin);
        auto a = ray.direction.dot(ray.direction);
        auto b = 2.0 * ray.direction.dot(oc);
        auto c = oc.dot(oc) - radius * radius;

        auto discriminant = b * b - 4 * a * c;

        if (discriminant < 0.0) {
            return;
        }

        // 与巨大球体求交的时候，会出现判别式大于0，但是有两个负根的情况，
        // 这种情况出现在射线方向的反向延长线能和球体相交的场合。
        auto t1 = (-b - std::sqrt(discriminant)) / (2 * a);
        auto t2 = (-b + std::sqrt(discriminant)) / (2 * a);

        auto position1 = ray.position(t1);

        auto normal1 = normalAt(position1);

        auto position2 = ray.position(t2);

        auto normal2 = normalAt(position2);

        if ((t1 > 0.0) || (t2 > 0.0)) {
            intersections[0] = { true, false, 1, t1, this, position1, normal1, ray };
            intersections[1] = { true, false, 1, t2, this, position2, normal2, ray };
        }
    }

    Tuple origin;
    double radius;
};

#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

CUDA_DEVICE void writePixel(Tuple* pixelBuffer, int32_t index, const Tuple& pixelColor) {
    pixelBuffer[index] = pixelColor;
}

CUDA_GLOBAL void createObject(Shape** object, Tuple origin, double radius) {
    // It is necessary to create object representing a function
    // directly in global memory of the GPU device for virtual
    // functions to work correctly, i.e. virtual function table
    // HAS to be on GPU as well.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        //auto* t = new Test();
        //t->foo();
        (*object) = new Sphere(origin, radius);
    }
}

CUDA_GLOBAL void deleteObject(Shape** object) {
    delete (*object);
}

CUDA_GLOBAL void fillBufferKernel(int32_t width, int32_t height, Payload* payload) {
    int32_t row = threadIdx.y + blockIdx.y * blockDim.y;
    int32_t column = threadIdx.x + blockIdx.x * blockDim.x;
    int32_t index = row * width + column;

    //auto viewport = payload->viewport;

    Tuple defaultColor = Color::skyBlue;
    Tuple pixelColor = defaultColor;

    const int32_t samplesPerPixel = 8;

    for (int i = 0; i < samplesPerPixel; i++) {
        hiprandState state;
        hiprand_init((unsigned long long)clock() + column, 0, 0, &state);

        double rx = hiprand_uniform_double(&state);
        double ry = hiprand_uniform_double(&state);

        //auto x = (viewport->height * (column + 0.5 + rx) / width - 1) * viewport->imageAspectRatio * viewport->scale;
        //auto y = (1.0 - viewport->height * (row + 0.5 + ry) / height) * viewport->scale;
        auto x = (static_cast<double>(column) + rx) / (width - 1);
        auto y = (static_cast<double>(row) + ry) / (height - 1);
        
        auto ray = payload->camera->getRay(x, y);

        Intersection intersections[MAXELEMENTS];

        int32_t count = 0;
    
        //payload->world->intersect(ray, intersections, &count);
        //payload->world->foo(ray, &count);
        //payload->world->getObject(0)->intersect(ray, intersections);
        Shape* object = new Sphere();

        object->intersect(ray, intersections);

        //auto hit = nearestHit(intersections, count);

        //if (hit.bHit) {
        //    pixelColor += hit.normal;
        //}
        //else {
        //    pixelColor += defaultColor;
        //}
    }

    writePixel(payload->pixelBuffer, index, pixelColor / samplesPerPixel);
}

void fillBufferCuda();

void queryDeviceProperties() {
    int32_t deviceIndex = 0;
    hipDeviceProp_t devicePro;
    hipGetDeviceProperties(&devicePro, deviceIndex);

    std::cout << "使用的GPU device：" << deviceIndex << ": " << devicePro.name << std::endl;
    std::cout << "SM的数量：" << devicePro.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devicePro.sharedMemPerBlock / 1024.0 << "KB\n";
    std::cout << "每个SM的最大线程块数：" << devicePro.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "每个线程块的最大线程数：" << devicePro.maxThreadsPerBlock << std::endl;
    std::cout << "每个SM的最大线程数：" << devicePro.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个SM的最大线程束数：" << devicePro.warpSize << std::endl;
}

int main()
{
    queryDeviceProperties();

    fillBufferCuda();

    return 0;
}

void fillBufferCuda() {
    // Choose which GPU to run on, change this on a multi-GPU system.
    gpuErrorCheck(hipSetDevice(0));

    constexpr auto width = 640;
    constexpr auto height = 480;

#if 1
    Payload* payload = nullptr;

    gpuErrorCheck(hipMallocManaged((void**)&payload, sizeof(Payload)));

    gpuErrorCheck(hipMallocManaged((void**)&payload->pixelBuffer, width * height * sizeof(Tuple)));

    gpuErrorCheck(hipMallocManaged((void**)&payload->viewport, sizeof(Viewport)));

    payload->viewport->fov = 90.0;
    payload->viewport->scale = std::tan(Math::radians(payload->viewport->fov / 2));

    payload->viewport->imageAspectRatio = static_cast<double>(width) / height;

    payload->viewport->height = 2.0 * payload->viewport->scale;
    payload->viewport->width = payload->viewport->height * payload->viewport->imageAspectRatio;

    gpuErrorCheck(hipMallocManaged((void**)&payload->camera, sizeof(Camera)));

    payload->camera->init(width, height);
    payload->camera->computeParameters();

    Shape** objects[2];

    gpuErrorCheck(hipMallocManaged((void**)&objects[0], sizeof(Shape**)));
    gpuErrorCheck(hipMallocManaged((void**)&objects[1], sizeof(Shape**)));

    createObject<<<1, 1>>>(objects[0], point(-2.0, 0.0, -3.0), 1.0);
    createObject<<<1, 1>>>(objects[1], point(2.0, 0.0, -3.0), 1.0);
    
    gpuErrorCheck(hipDeviceSynchronize());
    
    gpuErrorCheck(hipMallocManaged((void**)&payload->world, sizeof(World)));

    payload->world->addObject(*objects[0]);
    payload->world->addObject(*objects[1]);

    payload->objectCount = 2;

    //gpuErrorCheck(hipMallocManaged((void**)&payload->object->material, sizeof(Material)));

    gpuErrorCheck(hipMallocManaged((void**)&payload->intersections, sizeof(Intersection) * 2));
  
    Timer timer;

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    fillBufferKernel<<<gridSize, blockSize>>>(width, height, payload);

    gpuErrorCheck(hipDeviceSynchronize());

    timer.stop();

    writeToPPM("render.ppm", width, height, payload->pixelBuffer);

    gpuErrorCheck(hipFree(payload->intersections));
    deleteObject<<<1, 1>>>(objects[0]);
    deleteObject<<<1, 1>>>(objects[1]);
    gpuErrorCheck(hipDeviceSynchronize());
    //gpuErrorCheck(hipFree(payload->object));
    gpuErrorCheck(hipFree(payload->viewport));
    gpuErrorCheck(hipFree(payload->pixelBuffer));
#endif
}