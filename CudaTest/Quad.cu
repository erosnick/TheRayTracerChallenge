#include "hip/hip_runtime.h"
#include "Quad.h"
#include "Triangle.h"
#include "Intersection.h"

CUDA_HOST_DEVICE Quad::Quad(char* inName)
: name(inName) {
    auto triangle = new Triangle(point(-1.0, 0.0, -1.0), point(-1.0, 0.0, 1.0), point(1.0, 0.0, 1.0));
    triangles[0] = triangle;

    triangle = new Triangle(point(-1.0, 0.0, -1.0), point(1.0, 0.0, 1.0), point(1.0, 0.0, -1.0));
    triangles[1] = triangle;
}

CUDA_HOST_DEVICE void Quad::setTransformation(const Matrix4& inTransformation, bool bTransformPosition) {
    Shape::setTransformation(inTransformation);

    for (auto& triangle : triangles) {
        triangle->setTransformation(inTransformation);
    }
}

CUDA_HOST_DEVICE void Quad::transform(const Matrix4& inTransformation) {
    Shape::transform(inTransformation);

    for (auto& triangle : triangles) {
        triangle->transform(inTransformation);
    }
}

CUDA_HOST_DEVICE void Quad::transformNormal(const Matrix4& worldMatrix) {
    for (auto& triangle : triangles) {
        triangle->transformNormal(worldMatrix);
    }
}

CUDA_HOST_DEVICE Tuple Quad::normalAt(const Tuple& position) const {
    return triangles[0]->normalAt(position);
}

CUDA_HOST_DEVICE bool Quad::intersect(const Ray& ray, Intersection* intersections) {
    Intersection intersections1;
    
    triangles[0]->intersect(ray, &intersections1);
    Intersection intersections2;
    
    if (intersections1.bHit) {
        triangles[1]->intersect(ray, &intersections2);
    }

    int32_t count = 0;

    // ������ı�������һ���������ཻ������Ϊ�ཻ����Ϊ���������ι���
    if (intersections1.bHit) {
        intersections[0] = intersections1;
        count++;
    }
    else if (intersections2.bHit) {
        intersections[0] = intersections2;
        count++;
    }

    // ���˵�Quad����ʹ��(����Cube�Ĳ���)ʱt < 0�����
    if (count > 0) {
        intersections[0].object = this;

        if (!bCube && intersections[0].t < Math::epsilon) {
            //result.clear();
        }
    }

    return count > 0;
}

bool Quad::onQuad(const Tuple& inPosition, Tuple& normal) {
    auto p0p1 = (inPosition - triangles[0]->v0).normalize();
    auto p0p2 = (inPosition - triangles[1]->v2).normalize();

    auto normal1 = triangles[0]->normalAt(inPosition);
    auto normal2 = triangles[1]->normalAt(inPosition);
    
    // ����Ҫ�þ���ֵ����Ϊ�����ֵ����Ϊ��(�۽ǵ����)
    if (std::abs(p0p1.dot(normal1)) <= Math::epsilon) {
        normal = normal1;
    }
    else if (std::abs(p0p2.dot(normal2)) <= Math::epsilon) {
        normal = normal2;
    }

    return (normal != vector(0.0));
}

void Quad::setMaterial(Material* inMaterial) {
    material = inMaterial;
    for (auto& triangle : triangles) {
        triangle->setMaterial(inMaterial);
    }
}
