#include "hip/hip_runtime.h"
#include "Cube.h"
#include "Intersection.h"
#include "Quad.h"

#include <algorithm>

Cube::Cube() {
    initQuads();
}

void Cube::setTransformation(const Matrix4& inTransformation, bool bTransformPosition) {
    Shape::setTransformation(inTransformation, bTransformPosition);

    for (auto& quad : quads) {
        quad->setTransformation(inTransformation);
    }
}

void Cube::transform(const Matrix4& inTransformation) {
    Shape::transform(inTransformation);

    for (auto& quad : quads) {
        quad->transform(inTransformation);
    }
}

CUDA_HOST_DEVICE void Cube::updateTransformation() {
    for (auto& quad : quads) {
        quad->updateTransformation();
    }
}

void Cube::transformNormal(const Matrix4& worldMatrix) {
    for (auto& quad : quads) {
        quad->transformNormal(worldMatrix);
    }
}

Tuple Cube::normalAt(const Tuple& position) const {
    auto normal = vector(0.0);
    for (const auto& quad : quads) {
        if (quad->onQuad(position, normal)) {
            // ������任���ߣ��ᵼ�¶��߳�����������㣿
            // ���������ַ��ط��ߵķ�ʽ֮����ڲ��죬���Ų�
            return normal;
        }
    }

    return normal;
}

bool Cube::intersect(const Ray& ray, Array<Intersection>& intersections) {
    for (const auto& quad : quads) {
        if (quad->intersect(ray, intersections)) {
            auto& result = intersections.last();
            result.subObject = result.object;
            result.object = this;
        }
    }

    auto bHit = intersections.size();

    if (bHit) {
        if (intersections.size() == 1 && intersections[0].t < Math::epsilon) {
            intersections.remove(intersections.last());
        }
        else {
            auto hit = nearestHit(intersections);
            normal = hit.subObject->normalAt();
        }
    }

    return bHit;
}

bool Cube::intersect(const Ray& ray, Intersection intersections[]) {
    auto count = 0;
    for (const auto& quad : quads) {
        if (quad->intersect(ray, intersections)) {
            auto& result = intersections[count];
            result.subObject = result.object;
            result.object = this;
            count++;

            if (count == 2) {
                break;
            }
        }
    }

    auto bHit = count > 0;

    if (bHit) {
        if (count == 1 && intersections[0].t < Math::epsilon) {
            intersections[0] = Intersection();
        }
        else {
            auto hit = nearestHit(intersections, count);
            normal = hit.subObject->normalAt();
        }
    }

    return bHit;
}

//std::tuple<double, double> Cube::checkAxis(double origin, double direction) {
//    auto tminNumerator = -1.0 - origin;
//    auto tmaxNumerator = 1.0 - origin;
//
//    auto tmin = Math::infinityd;
//    auto tmax = Math::infinityd;
//
//    if (std::abs(direction) >= Math::epsilon) {
//        tmin = tminNumerator / direction;
//        tmax = tmaxNumerator / direction;
//    }
//    else {
//        tmin = tminNumerator * Math::infinityd;
//        tmax = tmaxNumerator * Math::infinityd;
//    }
//    if (tmin > tmax) {
//        std::swap(tmin, tmax);
//    }
//
//    return { tmin, tmax };
//}

void Cube::initQuads() {
    auto top = new Quad("Top", true);
    top->setTransformation(translate(0.0, 1.0, 0.0));

    quads.add(top);

    auto bottom = new Quad("Bottom", true);
    bottom->transformNormal(rotateX(Math::pi));
    bottom->setTransformation(translate(0.0, -1.0, 0.0) * rotateX(Math::pi));

    quads.add(bottom);

    auto back = new Quad("Back", true);
    back->transformNormal(rotateX(-Math::pi_2));
    back->setTransformation(translate(0.0, 0.0, -1.0) * rotateX(-Math::pi_2));

    quads.add(back);

    auto front = new Quad("Front", true);
    front->transformNormal(rotateX(Math::pi_2));
    front->setTransformation(translate(0.0, 0.0, 1.0) * rotateX(Math::pi_2));

    quads.add(front);

    auto left = new Quad("Left", true);
    left->transformNormal(rotateZ(Math::pi_2));
    left->setTransformation(translate(-1.0, 0.0, 0.0) * rotateZ(Math::pi_2));

    quads.add(left);

    auto right = new Quad("Right", true);
    right->transformNormal(rotateZ(-Math::pi_2));
    right->setTransformation(translate(1.0, 0.0, 0.0) * rotateZ(-Math::pi_2));

    quads.add(right);
}

void Cube::setMaterial(Material* inMaterial) {
    material = inMaterial;
    for (auto& quad : quads) {
        quad->setMaterial(inMaterial);
    }
}

void Cube::setMaterial(Material* inMaterial, int32_t quadIndex) {
    if (quadIndex > quads.size() - 1) {
        return;
    }

    quads[quadIndex]->setMaterial(inMaterial);
}
